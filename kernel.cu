#include "hip/hip_runtime.h"
#include "graph.h"
#include "cudahelper.h"

int main()
{
    clock_t beg,end;
    beg=clock();
    int source = 0;
    int dest = 0;
    hipError_t err = hipSuccess;
    
    size_t size = numberOfVertex  * sizeof(float);

    float *sumOfOutDegree=new float[numberOfVertex ];

    //Allocate the device memory
    float *d_Sum_Of_Degree = NULL;
    hipMalloc((void **)&d_Sum_Of_Degree, size);
    if(d_Sum_Of_Degree == NULL)
    {
        cout << "Failed"<<endl;
    }

    float *d_PR = NULL;
    hipMalloc((void**)&d_PR,size);
    if (d_PR == NULL)
    {
        cout << "Failed" << endl;
    }

    float *d_Graph = NULL;
    
    hipMalloc((void **)&d_Graph, size * numberOfVertex );
    if (d_Graph == NULL)
    {
        cout <<"Failed" << endl;
    }
	
    //Read Graph file.
    fstream fp("f1.txt",ios::in);
    if(!fp.is_open())
    {
        printf("Failed to open file.\n");
    }

    //host memory allocate

    float *Grap=new float[numberOfVertex*numberOfVertex ];
	
    float *PR=new float[numberOfVertex ];
    float *PR_Temp=new float[numberOfVertex];


    //init
    for (int i = 0; i < numberOfVertex ; ++i)
    {
        PR[i] = InitPageRankValue;
        PR_Temp[i] = InitPageRankValue;
    }

    for (int i = 0; i < numberOfVertex ; ++i)
    {
        for (int j = 0; j < numberOfVertex ; ++j)
        {
            Grap[i*numberOfVertex+j] = 0;
        }
    }

    int edge = 0;
	
    //read from Graph.txt
    while (!fp.eof())
    {
        fp >> source >> dest;
        Grap[(source-1)*numberOfVertex+(dest-1)] = 1;
	Grap[(dest-1)*numberOfVertex+(source-1)] = 1;
	edge++;
    }
    printf("Graph build Done!\n");

    //copy
    err = hipMemcpy(d_Graph, Grap, numberOfVertex *size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //invoke PageRank.
    //CPU Routine
    for (int i = 0; i < numberOfVertex; ++i)
    {
        PR[i] = InitPageRankValue;
    }

    int iter = 0;
    for (int m = 0; m < Max_Iteration_Number; ++m)
    {

        iter ++;

        //CUDA event timing
        
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        
        //calculate sum of out degree

	claculateSumOfOutDegree<<<256,256>>>(d_Sum_Of_Degree, d_Graph);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        //copy
        err = hipMemcpy(d_PR, PR, size, hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

	// add the sum
	PRAdd<<<256,256>>>(d_PR, d_Graph, d_Sum_Of_Degree);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);

        hipMemcpy(PR_Temp, d_PR, size, hipMemcpyDeviceToHost);

       if (END(PR_Temp, PR))
        {
            break;
        }
        else
	{
            for (int i = 0; i < numberOfVertex ; ++i)
            {
                PR[i] = PR_Temp[i];
            }
        }	
    }

    hipMemcpy(d_PR, PR, size, hipMemcpyHostToDevice);    

    fp.close();
    string ip,tok;
    fp.open("genes");
    map<string,int>gen;
    
    while(getline(fp,ip))
    {
	istringstream iss(ip);
	iss>>source>>tok;
	gen.insert(pair<string,int>(tok,source-1));
    }
	
   fp.close();
   fp.open("da.tsv");
   map<string,int>::iterator it;
	

   vector<int> v;
   long long t=0;
   while(getline(fp,ip))
   {
	t++;
	istringstream iss(ip);
	iss>>tok;
	it=gen.find(tok);
	if(it!=gen.end())
	{
		v.push_back(gen.find(tok)->second);
	}	
   }

   printf("\n size:%lld",t);
   int *va = new int[v.size()];
   copy(v.begin(),v.end(),va);
	
   int *dva=0;
   hipMalloc((void **)&dva,sizeof(int)*v.size());
   err = hipMemcpy(dva, va, sizeof(int)*v.size(), hipMemcpyHostToDevice);

   if (err != hipSuccess)
   {
   	fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int *dfreq=0;
    float *dans=0;
    hipMalloc((void **)&dfreq,sizeof(int)*numberOfVertex);
    hipMalloc((void **)&dans,sizeof(float)*numberOfVertex);
		
    calcfreq<<<256,256>>>(dva,dfreq,v.size());
		
    mul<<<256,256>>>(dans,dfreq,d_PR);
		
   thresh<<<256,256>>>(d_Graph,dans,100);

		

   err = hipMemcpy(Grap, d_Graph, numberOfVertex *size,hipMemcpyDeviceToHost);

   if (err != hipSuccess)
   {
   	fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
   }

   freopen("subnets2.txt","w",stdout);
   Graph g(numberOfVertex);
   
   for(int i=0;i<numberOfVertex;i++)
   {
	for(int j=0;j<numberOfVertex;j++)
	{
		if(Grap[i*numberOfVertex+j]==1)
		g.addEdge(i,j);
	}			
   }	
	
  g.printSCCs();
				

  end=clock();
  printf("\n Time taken is : %.9f",(double)(end-beg)/CLOCKS_PER_SEC);
}
