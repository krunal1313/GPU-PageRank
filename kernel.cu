#include "hip/hip_runtime.h"
#include "graph.h"
#include "cudahelper.h"

int main()
{
    //char ch;
	clock_t beg,end;
	beg=clock();
    int source = 0;
    int dest = 0;
    hipError_t err = hipSuccess;
    
    size_t size = numberOfVertex  * sizeof(float);

    float *sumOfOutDegree=new float[numberOfVertex ];

    //Allocate the device memory
    float *d_Sum_Of_Degree = NULL;
    hipMalloc((void **)&d_Sum_Of_Degree, size);
    if(d_Sum_Of_Degree == NULL)
    {
        cout << "Failed"<<endl;
    }

    float *d_PR = NULL;
    hipMalloc((void**)&d_PR,size);
    if (d_PR == NULL)
    {
        cout << "Failed" << endl;
    }

    float *d_Graph = NULL;
    
	hipMalloc((void **)&d_Graph, size * numberOfVertex );
    if (d_Graph == NULL)
    {
        cout <<"Failed" << endl;
    }

    //thread number
/*
    int threadsPerBlock = numberOfVertex ;
    int blocksPerGrid =(numberOfVertex  + threadsPerBlock - 1) / threadsPerBlock;
	*/
    //Read Graph file.

    fstream fp("f1.txt",ios::in);
    if(!fp.is_open())
    {
        printf("Failed to open file.\n");
    }

    //output file
  /*  fstream prFile("PageRankValue.txt", ios::out);
    if (!prFile.is_open())
    {
        printf("Failed to open file PRV\n");
    }
	*/
    //host memory allocate

	float *Grap=new float[numberOfVertex*numberOfVertex ];
	
	float *PR=new float[numberOfVertex ];
    float *PR_Temp=new float[numberOfVertex];


    //init
    for (int i = 0; i < numberOfVertex ; ++i)
    {
        PR[i] = InitPageRankValue;
        PR_Temp[i] = InitPageRankValue;
    }

    for (int i = 0; i < numberOfVertex ; ++i)
    {
        for (int j = 0; j < numberOfVertex ; ++j)
        {
            Grap[i*numberOfVertex+j] = 0;
        }
    }
	int edge = 0;
    //read from Graph.txt
    while (!fp.eof()){

        fp >> source >> dest;
     //   std::cout << source << ' '<< dest << std::endl;

        Grap[(source-1)*numberOfVertex+(dest-1)] = 1;
		Grap[(dest-1)*numberOfVertex+(source-1)] = 1;
		edge++;
    }
    printf("Graph build Done!\n");
//    printf("----------------------------------------------------------\n");


    //copy
    err = hipMemcpy(d_Graph, Grap, numberOfVertex *size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //invoke PageRank.
	//CPU Routine
 
  //  int CPUTime  = 0;
//    CPUTime = PageRank(Graph, PR);


	for (int i = 0; i < numberOfVertex; ++i)
    {
        PR[i] = InitPageRankValue;
    }
   // printf("--------------------------------------------------------\n");
  //  clock_t begin, end;
    int iter = 0;
//    float SumOfGPUTime = 0;
    //begin = clock();
    for (int m = 0; m < Max_Iteration_Number; ++m)
    {
        /*
        for (int i = 0; i < numberOfVertex ; ++i)
        {
            for (int j = 0; j < numberOfVertex ; ++j)
            {
                printf("%f\t", Graph[i][j]);
            }
            printf("\n");
        }
        */

        iter ++;

        //CUDA event timing
        
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        
        //calculate sum of out degree

        //claculateSumOfOutDegree<<<blocksPerGrid, threadsPerBlock>>>(d_Sum_Of_Degree, d_Graph);
		claculateSumOfOutDegree<<<256,256>>>(d_Sum_Of_Degree, d_Graph);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        /*
        hipMemcpy(sumOfOutDegree, d_Sum_Of_Degree, size, hipMemcpyDeviceToHost);
        for (int i = 0; i < numberOfVertex ; ++i)
        {
            cout << sumOfOutDegree[i] <<'\t';
        }
        */

        //copy
        err = hipMemcpy(d_PR, PR, size, hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        //PRAdd<<<blocksPerGrid, threadsPerBlock>>>(d_PR, d_Graph, d_Sum_Of_Degree);
		PRAdd<<<256,256>>>(d_PR, d_Graph, d_Sum_Of_Degree);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);

     //   SumOfGPUTime += elapsedTime;

        hipMemcpy(PR_Temp, d_PR, size, hipMemcpyDeviceToHost);

       if (END(PR_Temp, PR))
        {
            break;
        }
        else{
            for (int i = 0; i < numberOfVertex ; ++i)
            {
                PR[i] = PR_Temp[i];
            }
        }
        
		
    }
	hipMemcpy(d_PR, PR, size, hipMemcpyHostToDevice);
    //end = clock();
    


    //printf("%d\n", vertex);
    
  /*  for (int i = 0; i < numberOfVertex; ++i)
    {
        prFile << i <<" "<< PR[i] << endl;
    }
    */

    

	//cout<<PR[9204];
    fp.close();
	string ip,tok;
	fp.open("genes");
	map<string,int>gen;
	while(getline(fp,ip))
	{
		istringstream iss(ip);
		iss>>source>>tok;
		gen.insert(pair<string,int>(tok,source-1));
	}
	
	fp.close();
	fp.open("da.tsv");
	map<string,int>::iterator it;
	

	vector<int>v;
	long long t=0;
	while(getline(fp,ip))
	{
		t++;
		istringstream iss(ip);
		iss>>tok;
		it=gen.find(tok);
		if(it!=gen.end())
		{
			//freq[gen.find(tok)->second]++;
			v.push_back(gen.find(tok)->second);
		}	
	}
	printf("\n size:%lld",t);
    int *va=new int[v.size()];
	copy(v.begin(),v.end(),va);
	
	int *dva=0;
	hipMalloc((void **)&dva,sizeof(int)*v.size());
	err = hipMemcpy(dva, va, sizeof(int)*v.size(), hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

		int *dfreq=0;
		float *dans=0;
		hipMalloc((void **)&dfreq,sizeof(int)*numberOfVertex);
		hipMalloc((void **)&dans,sizeof(float)*numberOfVertex);
		
		calcfreq<<<256,256>>>(dva,dfreq,v.size());
		
		mul<<<256,256>>>(dans,dfreq,d_PR);
		
	/*	float *ans=new float[numberOfVertex];

		err = hipMemcpy(ans, dans, size,hipMemcpyDeviceToHost);
		ofstream l;
		l.open("final.txt");
		for(int i=0;i<numberOfVertex;i++)
		{
			l<<ans[i]<<endl;
		}
		l.close();
		*/
		thresh<<<256,256>>>(d_Graph,dans,100);

		

		err = hipMemcpy(Grap, d_Graph, numberOfVertex *size,hipMemcpyDeviceToHost);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

		
/*		int cnt;
		ofstream op;
		op.open("result.txt");
		vector<int>vi;
		for(int i=0;i<numberOfVertex;i++)
		{
			cnt=0;
			vi.clear();
			
			for(int j=0;j<numberOfVertex;j++)
			{
				if(Grap[i*numberOfVertex+j]==1)
				{
					cnt++;
					vi.push_back(j);
				}
				
			}
			if(cnt>=3)
			{
				op<<i<<" : ";
				for(int k=0;k<vi.size();k++)
				op<<vi[k]<<" ";
				
				op<<endl;
			}
		 } 
*/

			freopen("subnets2.txt","w",stdout);
		Graph g(numberOfVertex);
		for(int i=0;i<numberOfVertex;i++)
			{
				for(int j=0;j<numberOfVertex;j++)
				{
					if(Grap[i*numberOfVertex+j]==1)
						g.addEdge(i,j);
					
				}			
			}	
			g.printSCCs();
				

		end=clock();
		printf("\n Time taken is : %.9f",(double)(end-beg)/CLOCKS_PER_SEC);


		getchar();
}
